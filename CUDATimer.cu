#include "CUDATimer.cuh"


CUDATimer::CUDATimer() {
    // TODO: some way of specifying EventCreate flags
    hipEventCreate(&ev_start);
    hipEventCreate(&ev_stop);
}


CUDATimer::~CUDATimer() {
    hipEventDestroy(ev_start);
    hipEventDestroy(ev_stop);
}


void CUDATimer::start(hipStream_t stream) {
    hipEventRecord(ev_start, stream);
}


float CUDATimer::get_elapsed_time_nosync(hipStream_t stream) {
    float et;
    hipEventRecord(ev_stop, stream);
    hipEventElapsedTime(&et, ev_start, ev_stop);
    return et;
}

float CUDATimer::get_elapsed_time_sync(hipStream_t stream) {
    float et;
    hipEventRecord(ev_stop, stream);
    hipEventSynchronize(ev_stop);
    hipEventElapsedTime(&et, ev_start, ev_stop);
    return et;
}

