#include <iostream>
#include <string>

#include <hip/hip_runtime.h>


void printDesc(const std::string &label, const hipChannelFormatDesc &desc) {
    std::cout << label << " -----------" << std::endl;
    std::cout << "x: " << desc.x << std::endl;
    std::cout << "y: " << desc.y << std::endl;
    std::cout << "z: " << desc.z << std::endl;
    std::cout << "w: " << desc.w << std::endl;
}

int main(int argc, char const *argv[])
{
    hipChannelFormatDesc desc;

    desc = hipCreateChannelDesc<float4>();
    printDesc("float4", desc);

    desc = hipCreateChannelDesc<float>();
    printDesc("float", desc);

    desc = hipCreateChannelDesc<int>();
    printDesc("int", desc);

    desc = hipCreateChannelDesc<char>();
    printDesc("char", desc);

    desc = hipCreateChannelDesc<char4>();
    printDesc("char4", desc);

    return 0;
}
